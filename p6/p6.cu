//Nicholas Hodge


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#define CUDA_CALL(x){ \
	if ( (x) != hipSuccess ) { \
		printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
		printf("  %s\n", hipGetErrorString(hipGetLastError())); \
		exit(EXIT_FAILURE); \
	} \
}

__global__
void func(double *result, double *input, double *weights, int in){
	
	float calc = 0.0;
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	//apply weights to inputs
	for(int i = 0; i < in; i++){

		calc += input[i] * weights[id + i];

	}

	calc = (1.0)/(1.0 + exp(-calc));

	result[id] = calc;


}


int main(int argc, char *argv[]){

	int fd;
	double fl;
	double *result, *input, *weights;
	double *dev_results, *dev_input, *dev_weights;
	int rc, numBlocks, numThreads, numInputs, numNeurons, numWeights;


	fd = open(argv[1], O_RDONLY);
	if(fd < 0){

		printf("**** failed to open file :%s:\n", argv[1]);
		exit(-1);

	}


	//get numNeurons and numInputs
	rc = read(fd, &fl, sizeof(double));
	numNeurons = (int) fl;
	rc = read(fd, &fl, sizeof(double));
	numInputs = (int) fl;

	//get inputs
	input = (double *)malloc(numInputs * sizeof(double));
	rc = read(fd, input, numInputs * sizeof(double));

	//get weights
	numWeights = numInputs * numNeurons;
	weights = (double *)malloc(numWeights * sizeof(double));
	rc = read(fd, weights, numWeights * sizeof(double));

	close(fd);

	if((numInputs * numNeurons) < 32){

                numThreads = numInputs * numNeurons;

        }else{

                numThreads = 32;

        }


	numBlocks = numWeights/numThreads;
	if(numBlocks == 0){
		numBlocks = 1;
	}

	result = (double *)malloc(numNeurons * sizeof(double));

	CUDA_CALL(hipMalloc((void**) &dev_results, numNeurons * sizeof(double)));
	CUDA_CALL(hipMalloc((void**) &dev_input, numInputs * sizeof(double)));
	CUDA_CALL(hipMalloc((void**) &dev_weights, numWeights * sizeof(double)));


	CUDA_CALL(hipMemcpy(dev_results, result, numNeurons * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_input, input, numInputs * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dev_weights, weights, numWeights * sizeof(double), hipMemcpyHostToDevice));


	func<<<numBlocks, numThreads>>>(dev_results, dev_input, dev_weights, numInputs);


	CUDA_CALL(hipMemcpy(result, dev_results, numNeurons * sizeof(double), hipMemcpyDeviceToHost));

	

	if(argc < 3){

		for(int i = 0; i < numNeurons; i++){

			printf("%d  %f\n", i, result[i]);

		}
	}



	double total = 0;
	for(int i = 0; i < numNeurons; i++){

		total+= result[i];

	}

	printf("total %f\n", total);

	free(input);
	free(weights);
	free(result);
	hipFree(dev_results);
	hipFree(dev_input);
	hipFree(dev_weights);

}
